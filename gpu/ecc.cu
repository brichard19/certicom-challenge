#include "hip/hip_runtime.h"
#include "ecmath.cuh"
#include "ecc.cuh"

#define get_global_id() (blockDim.x * blockIdx.x + threadIdx.x)

#define get_global_size() (blockDim.x * gridDim.x)

__device__ void print_big_int(uint131_t& x)
{
  printf("%.8X%.16lX%.16lX", x.w.v2, x.w.v1, x.w.v0);
}


__device__ int get_bit(uint131_t x, int bit)
{
  if(bit >= 128) {
    return (x.w.v2 >> (bit & 0x7f)) & 1;
  } else if(bit >= 64) {
    return (x.w.v1 >> (bit & 0x3f)) & 1;
  } else {
    return (x.w.v0 >> (bit & 0x1f)) & 1;
  }
}

__device__ uint131_t load(const uint131_vec_ptr_t p, int n, int i)
{
  uint131_t value;

  __int128* p128 = (__int128*)p;
  uint32_t* p32 = (uint32_t*)p;
  
  __int128 lo = p128[i];
  uint32_t hi = p32[n * 4 + i];

  value.w.v0 = (uint64_t)lo;
  value.w.v1 = (uint64_t)(lo >> 64);
  value.w.v2 = hi;

  return value;
}

__device__ void store(uint131_vec_ptr_t p, int n, int i, uint131_t x)
{
  __int128* p128 = (__int128*)p;
  uint32_t* p32 = (uint32_t*)p;

  __int128 lo = (__int128)x.w.v1 << 64 | x.w.v0;
  p128[i] = lo;

  p32[4 * n + i] = x.w.v2;
}

// If the private key bit for P is 1, then add Q to P
template<int CURVE> __device__ void do_step_impl(uint131_vec_ptr_t global_px, uint131_vec_ptr_t global_py,
                                   uint131_vec_ptr_t global_rx, uint131_vec_ptr_t global_ry,
                                   uint131_vec_ptr_t mbuf, int count,
                                   DPResult* result, int* result_count,
                                   StagingPoint* staging, int* staging_count,
                                   uint131_t* priv_key_a,
                                   uint64_t counter,
                                   uint64_t* start_pos,
                                   uint64_t dpmask)
{
  const int rmask = 0x1f;
  const int gid = get_global_id();
  const int dim = get_global_size();

  int i = gid;

  uint131_t inverse;

  // Perform Qx - Px and then multiply them together
  for(; i < count; i+=dim) {
    uint131_t px = load(global_px, count, i);

    // TODO: Proper mask
    int idx = px.w.v0 & rmask;

    uint131_t rx = load(global_rx, 32, idx);

    // Point addition, rx - px
    uint131_t t = sub<CURVE>(rx, px);
    if(i > gid) {
      inverse = mul<CURVE>(inverse, t);
    } else {
      inverse = t;
    }
    store(mbuf, count, i, inverse);
  }

  // Perform inversion
  inverse = inv<CURVE>(inverse);

  // Start at last element (undo final loop counter add)
  i -= dim;

  // Complete addition

  for(; i >= gid; i-=dim) {
    uint131_t px = load(global_px, count, i);
    uint131_t py = load(global_py, count, i);

    int idx = px.w.v0 & rmask;

    uint131_t rx = load(global_rx, 32, idx);
    uint131_t ry = load(global_ry, 32, idx);

    uint131_t s;

    if(i > gid) {

      // Get the 2nd-last element (product of all factors up to that number)
      // e.g. abcd
      uint131_t m = load(mbuf, count, i - dim);

      // Multiply to cancel out all factors except the last one
      // e.g. abcd * (abcde)^-1 = e^-1
      s = mul<CURVE>(inverse, m);

      // Cancel out from the inverse
      // e.g. abcde * e^-1 = abcd
      uint131_t diff = sub<CURVE>(rx, px);
      
      inverse = mul<CURVE>(inverse, diff);

    } else {
      s = inverse;
    }

    // Perform point addition
    uint131_t rise = sub<CURVE>(ry, py);
    s = mul<CURVE>(s, rise);
    uint131_t s2 = square<CURVE>(s);

    uint131_t tmp1 = sub<CURVE>(s2, px);
    uint131_t x = sub<CURVE>(tmp1, rx);

    uint131_t tmp2 = sub<CURVE>(px, x);
    uint131_t tmp3 = mul<CURVE>(s, tmp2);
    uint131_t y = sub<CURVE>(tmp3, py);

    store(global_px, count, i, x);
    store(global_py, count, i, y);
  }
}




// If the private key bit for P is 1, then add Q to P
template<int CURVE> __device__ void batch_multiply_step(uint131_vec_ptr_t global_px, uint131_vec_ptr_t global_py, uint131_t* private_keys, uint131_t* global_qx, uint131_t* global_qy, uint131_vec_ptr_t mbuf, int priv_key_bit, int count)
{
  int gid = get_global_id();
  int dim = get_global_size();

  const uint131_t qx = global_qx[priv_key_bit];
  const uint131_t qy = global_qy[priv_key_bit];

  int i = gid;

  uint131_t one;
  if(CURVE == 131) {
    one = _p131_one;
  } else if(CURVE == 79) {
    one = _p79_one;
  }

  uint131_t inverse = one;
  
  // Perform Qx - Px and then multiply them together
  for(; i < count; i+=dim) {

    uint131_t px = load(global_px, count, i);
    uint131_t t;

    int bit = get_bit(private_keys[i], priv_key_bit);

    if(!bit || is_infinity(px) || equal(px, qx)) {
      
      // Nothing to do, just use 1
      t = one;
    } else {
      // Point addition, qx - px
      t = sub<CURVE>(qx, px);
    }

    inverse = mul<CURVE>(inverse, t);
    store(mbuf, count, i, inverse);
  }

  // Perform inversion
  inverse = inv<CURVE>(inverse);

  // Start at last element (undo final loop counter add)
  i -= dim;

  // Complete addition

  for(; i >= gid; i-=dim) {
    uint131_t px = load(global_px, count, i);
    uint131_t py = load(global_py, count, i);

    int bit = get_bit(private_keys[i], priv_key_bit);

    if(!bit) {
      continue;
    } else if(is_infinity(px)) {
      store(global_px, count, i, qx);
      store(global_py, count, i, qy);

      continue;
    } else if(equal(px, qx)) {
      store(global_px, count, i, load(global_qx, count, priv_key_bit + 1));
      store(global_py, count, i, load(global_qy, count, priv_key_bit + 1));
      continue;
    }
    
    uint131_t s;
    if(i > gid) {

      // Get the 2nd-last element (product of all factors up to that number)
      // e.g. abcd
      uint131_t m = load(mbuf, count, i - dim);

      // Multiply to cancel out all factors except the last one
      // e.g. abcd * (abcde)^-1 = e^-1
      s = mul<CURVE>(inverse, m);

      // Cancel out from the inverse
      // e.g. abcde * e^-1 = abcd
      uint131_t diff = sub<CURVE>(qx, px);
      
      inverse = mul<CURVE>(inverse, diff);

    } else {
      s = inverse;
    }

    // Perform point addition
    uint131_t rise = sub<CURVE>(qy, py);
    s = mul<CURVE>(s, rise);
    uint131_t s2 = square<CURVE>(s);

    uint131_t tmp1 = sub<CURVE>(s2, px);
    uint131_t x = sub<CURVE>(tmp1, qx);

    uint131_t tmp2 = sub<CURVE>(px, x);
    uint131_t tmp3 = mul<CURVE>(s, tmp2);
    uint131_t y = sub<CURVE>(tmp3, py);

    store(global_px, count, i, x);
    store(global_py, count, i, y);
  }
}

template<int CURVE> __device__ void sanity_check_impl(uint131_vec_ptr_t global_px, uint131_vec_ptr_t global_py, int count, int* errors)
{
  int gid = get_global_id();
  int dim = get_global_size();

  for(int i = gid; i < count; i += dim) {
    uint131_t x = load(global_px, count, i);
    uint131_t y = load(global_py, count, i);

    if(point_exists<CURVE>(x, y) == false) {
      atomicAdd(errors, 1);
    }
  }
}

__device__ void clear_public_keys_impl(uint131_vec_ptr_t x, uint131_vec_ptr_t y, int count)
{
  int idx = get_global_id();
  int dim = get_global_size();
 
  uint131_t point_zero;
  set_point_at_infinity(point_zero);
  for(int i = idx; i < count; i += dim) {
    store(x, count, i, point_zero);
  }
}

__device__ void reset_counters_impl(uint64_t* start_pos, uint64_t value, int count)
{
  int idx = get_global_id();
  int dim = get_global_size();
  
  for(int i = idx; i < count; i += dim) {
    start_pos[i] = value;
  }
}

// Set all public keys to point-at-infinity
extern "C" __global__ void clear_public_keys(uint131_vec_ptr_t x, uint131_vec_ptr_t y, int count)
{
  clear_public_keys_impl(x, y, count);
}

extern "C" __global__ void reset_counters(uint64_t* start_pos, uint64_t value, int count)
{
  reset_counters_impl(start_pos, value, count);
}



extern "C" __global__ void sanity_check_p131(uint131_vec_ptr_t global_px, uint131_vec_ptr_t global_py, int count, int* errors)
{
  sanity_check_impl<131>(global_px, global_py, count, errors);
}

extern "C" __global__ void sanity_check_p79(uint131_vec_ptr_t global_px, uint131_vec_ptr_t global_py, int count, int* errors)
{
  sanity_check_impl<79>(global_px, global_py, count, errors);
}


extern "C" __global__ void batch_multiply_p79(uint131_vec_ptr_t global_px, uint131_vec_ptr_t global_py, uint131_t* private_keys, uint131_vec_ptr_t mbuf, uint131_t* gx, uint131_t* gy, int priv_key_bit, int count)
{
    batch_multiply_step<79>(global_px, global_py, private_keys, gx, gy, mbuf, priv_key_bit, count);
}


extern "C" __global__ void do_step_p79(uint131_vec_ptr_t global_px, uint131_vec_ptr_t global_py,
                                   uint131_vec_ptr_t global_rx, uint131_vec_ptr_t global_ry,
                                   uint131_vec_ptr_t mbuf, int count,
                                   DPResult* result, int* result_count,
                                   StagingPoint* staging, int* staging_count,
                                   uint131_t* priv_key_a,
                                   uint64_t counter,
                                   uint64_t* start_pos,
                                   uint64_t dpmask)
{
  do_step_impl<79>(global_px, global_py, global_rx, global_ry, mbuf, count, result, result_count, staging, staging_count, priv_key_a, counter, start_pos, dpmask);
}



extern "C" __global__ void batch_multiply_p131(uint131_vec_ptr_t global_px, uint131_vec_ptr_t global_py, uint131_t* private_keys, uint131_vec_ptr_t mbuf, uint131_t* gx, uint131_t* gy, int priv_key_bit, int count)
{
    batch_multiply_step<131>(global_px, global_py, private_keys, gx, gy, mbuf, priv_key_bit, count);
}


extern "C" __global__ void check_for_dp(uint131_vec_ptr_t global_px, uint131_vec_ptr_t global_py,
                                   int count,
                                   DPResult* result, int* result_count,
                                   StagingPoint* staging, int* staging_count,
                                   uint131_t* priv_key_a,
                                   uint64_t counter,
                                   uint64_t* start_pos,
                                   uint64_t dpmask)
{
  const int gid = get_global_id();
  const int dim = get_global_size();

  int i = gid;

  for(; i < count; i+=dim) {
    uint131_t px = load(global_px, count, i);
    if(result != NULL && (px.w.v0 & dpmask) == 0) {
      // Record distinguished point
      int idx = atomicAdd(result_count, 1);

      DPResult r;

      r.a = priv_key_a[i];
      r.x = px;
      r.y = load(global_py, count, i);
      r.length = counter - start_pos[i];

      result[idx] = r;

      // Grab a new point from the staging buffer
      idx = atomicSub(staging_count, 1) - 1;
    
      uint131_t new_x = staging[idx].x;
      uint131_t new_y = staging[idx].y;
      priv_key_a[i] = staging[idx].a;

      start_pos[i] = counter;
      px = new_x;
      store(global_px, count, i, new_x);
      store(global_py, count, i, new_y);
    }
  }
}

extern "C" __global__ void do_step_p131(uint131_vec_ptr_t global_px, uint131_vec_ptr_t global_py,
                                   uint131_vec_ptr_t global_rx, uint131_vec_ptr_t global_ry,
                                   uint131_vec_ptr_t mbuf, int count,
                                   DPResult* result, int* result_count,
                                   StagingPoint* staging, int* staging_count,
                                   uint131_t* priv_key_a,
                                   uint64_t counter,
                                   uint64_t* start_pos,
                                   uint64_t dpmask)
{
  do_step_impl<131>(global_px, global_py, global_rx, global_ry, mbuf, count, result, result_count, staging, staging_count, priv_key_a, counter, start_pos, dpmask);
}